#include "hip/hip_runtime.h"
/*
 * CUDA MG solver with Unified Memory
 * Ruxi Qi @ UC Irvine, 2017
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <algorithm>
// For gdb
#include <signal.h>
// For timing
#include <sys/time.h>
// CUDA Runtime
#include <hip/hip_runtime.h>
// CUDA BLAS Library
#include "hipblas.h"
// For error handling and device pickup
#include "hip/hip_runtime_api.h"
#include "cuda_mg_wrapper.h"

// Global vairables
int l_xm;
int l_ym;
int l_zm;
int l_xmym;
int l_xmymzm;
int l_maxitn;
// For initialization
int l_l;
int l_m;
int l_n;
__device__ __managed__ int l_bcopt;
float l_accept;
//int mg_nlevel;
int ncyc_before;
int ncyc_after;
float l_pbkappa;
float l_epsout;
float l_h;
float l_wsor;
int l_itn;
float l_inorm;
float l_norm;

int threshold;

#define MG_NLEVEL 4
int mg_index[MG_NLEVEL + 1];
int mg_index_ext[MG_NLEVEL + 1];
int mg_x_idx[MG_NLEVEL + 1];
int mg_size[MG_NLEVEL][3];
float mg_onorm[MG_NLEVEL];

float *l_zv;
float *l_ad;
float *l_bv;
float *l_rv;
float *l_iv;
float *l_bz;
float *l_am1;
float *l_am2;
float *l_am3;
float *l_xv;

int devThreadsPerBlock = 8;

extern "C"
void init_param_c_(int *nx, int *ny, int *nz, int *p_maxitn, int *p_bcopt, float *p_accept, float *p_pbkappa, float *p_epsout, float *p_h, float *p_wsor) {
    l_xm = *nx;
    l_ym = *ny;
    l_zm = *nz;
    l_xmym = *nx * *ny;
    l_xmymzm = *nx * *ny * *nz;
    l_maxitn = *p_maxitn;
    l_bcopt = *p_bcopt;
    l_accept = *p_accept;
    ncyc_before = 10;
    ncyc_after = 10;
    l_pbkappa = *p_pbkappa;
    l_epsout = *p_epsout;
    l_h = *p_h;
    l_wsor = *p_wsor;

    threshold = 2;
}

extern "C"
void allocate_array_cuda_(int *solvopt) {
    if (!(*solvopt == 2 || *solvopt == 4)) {
        printf("Error: Only MG/SOR is supported now.\n");
        exit(2);
    }

    int m, l, n;

    // set indices for the finest level for all solvers
    mg_index_ext[0] = 0;
    mg_index[0] = 0;
    mg_x_idx[0] = 0;
    mg_size[0][0] = l_xm;
    mg_size[0][1] = l_ym;
    mg_size[0][2] = l_zm;
    m = l_xmymzm;
    l = m + l_xmym;
    n = l + l_xmym;

    // set indices for all other levels for MG only
    if (*solvopt == 2) {
        for (int i = 1; i < MG_NLEVEL; i++) {
            mg_index_ext[i] = l;
            mg_index[i] = m;
            mg_x_idx[i] = n;

            //l_bcopt != 10 for now
            for (int j = 0; j < 3; j++) {
                mg_size[i][j] = mg_size[i - 1][j] / 2;
            }
            m += mg_size[i][0] * mg_size[i][1] * mg_size[i][2];
            l += mg_size[i][0] * mg_size[i][1] * mg_size[i][2] + mg_size[i][0] * mg_size[i][1];
            n += mg_size[i][0] * mg_size[i][1] * mg_size[i][2] + 2 * mg_size[i][0] * mg_size[i][1];
        }

        mg_index_ext[MG_NLEVEL] = l;
        mg_index[MG_NLEVEL] = m;
        mg_x_idx[MG_NLEVEL] = n;
    }

    // Now for all arrays
    // Try __managed__ declaration later for performance tuning
    //__device__ __managed__ l_xv[n];
    // Note in Fortran these arrays index from 1, not 1-xmym etc.
    cudaErrorCheck(hipMallocManaged(&l_zv, sizeof(float) * m));
    cudaErrorCheck(hipMallocManaged(&l_ad, sizeof(float) * m));
    cudaErrorCheck(hipMallocManaged(&l_bv, sizeof(float) * m));
    cudaErrorCheck(hipMallocManaged(&l_rv, sizeof(float) * m));
    cudaErrorCheck(hipMallocManaged(&l_iv, sizeof(float) * m));
    cudaErrorCheck(hipMallocManaged(&l_bz, sizeof(float) * m));
    cudaErrorCheck(hipMallocManaged(&l_am1, sizeof(float) * l));
    cudaErrorCheck(hipMallocManaged(&l_am2, sizeof(float) * l));
    cudaErrorCheck(hipMallocManaged(&l_am3, sizeof(float) * l));
    cudaErrorCheck(hipMallocManaged(&l_xv, sizeof(float) * n));

    l_l = l;
    l_m = m;
    l_n = n;
}

extern "C"
void deallocate_array_cuda_() {
    hipFree(l_zv);
    hipFree(l_ad);
    hipFree(l_bv);
    hipFree(l_rv);
    hipFree(l_iv);
    hipFree(l_bz);
    hipFree(l_am1);
    hipFree(l_am2);
    hipFree(l_am3);
    hipFree(l_xv);
    hipDeviceReset();
}

extern "C"
void init_array_cuda_(int *solvopt, float *epsx, float *epsy, float *epsz, float *p_bv, float *p_iv, float *p_xs) {
    if (!(*solvopt == 2 || *solvopt == 4) ) {
        printf("Error: Only MG/SOR is supported now.\n");
        exit(2);
    }

    // Initialize arrays l_ad, l_am*, l_*v to 0 on device. Use 1D thread block.
    int blocksize = devThreadsPerBlock * devThreadsPerBlock * devThreadsPerBlock;

    // m
    int nblocks = (l_m - 1) / blocksize + 1;
    init_vector_kernel<<<nblocks, blocksize>>>(l_zv, l_m);
    init_vector_kernel<<<nblocks, blocksize>>>(l_ad, l_m);
    init_vector_kernel<<<nblocks, blocksize>>>(l_bv, l_m);
    init_vector_kernel<<<nblocks, blocksize>>>(l_rv, l_m);
    init_vector_kernel<<<nblocks, blocksize>>>(l_iv, l_m);
    cudaLaunchErrorCheck();

    // l
    nblocks = (l_l - 1) / blocksize + 1;
    init_vector_kernel<<<nblocks, blocksize>>>(l_am1, l_l);
    init_vector_kernel<<<nblocks, blocksize>>>(l_am2, l_l);
    init_vector_kernel<<<nblocks, blocksize>>>(l_am3, l_l);
    cudaLaunchErrorCheck();

    // n
    nblocks = (l_n - 1) / blocksize + 1;
    init_vector_kernel<<<nblocks, blocksize>>>(l_xv, l_n);
    cudaLaunchErrorCheck();

    // Ready to array assignment: do the easiest arrays first, directly copying from the caller
    // starting index passed from F90: l_xv(1:n) which covers 2*xmym buffer; p_xs(1:l_xmymzm) just core part
    // so l_xv(1+xmym:l_xmymzm+xmym), p_xs(1:l_xmymzm)
    // Eidt: copy to device memory to limit CPU page faults
    float *lp_bv, *lp_iv, *lp_xs;
    cudaErrorCheck(hipMalloc(&lp_bv, sizeof(float) * l_xmymzm));
    cudaErrorCheck(hipMalloc(&lp_iv, sizeof(float) * l_xmymzm));
    cudaErrorCheck(hipMalloc(&lp_xs, sizeof(float) * l_xmymzm));
    hipMemcpy(lp_bv, p_bv, l_xmymzm * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(lp_iv, p_iv, l_xmymzm * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(lp_xs, p_xs, l_xmymzm * sizeof(float), hipMemcpyHostToDevice);

    nblocks = (l_xmymzm  - 1) / blocksize + 1;
    copy_vector_kernel<<<nblocks, blocksize>>>(l_xv + l_xmym, lp_xs, l_xmymzm);
    cudaLaunchErrorCheck();

    copy_vector_kernel<<<nblocks, blocksize>>>(l_bv, lp_bv, l_xmymzm);
    cudaLaunchErrorCheck();

    // Set up local eps arrays for data assignment on the kernel
    int m = 0;
    for (int i = 0; i < MG_NLEVEL; i++) {
        m += (mg_size[i][0] + 1) * (mg_size[i][1] + 1) * (mg_size[i][2] + 1);
    }

    float *lepsx, *lepsy, *lepsz;
    cudaErrorCheck(hipMallocManaged(&lepsx, sizeof(float) * m));
    cudaErrorCheck(hipMallocManaged(&lepsy, sizeof(float) * m));
    cudaErrorCheck(hipMallocManaged(&lepsz, sizeof(float) * m));
    float *epsx_f, *epsy_f, *epsz_f;
    cudaErrorCheck(hipMalloc(&epsx_f, (l_xmymzm + l_ym * l_zm) * sizeof(float)));
    cudaErrorCheck(hipMalloc(&epsy_f, (l_xmymzm + l_xm * l_zm) * sizeof(float)));
    cudaErrorCheck(hipMalloc(&epsz_f, (l_xmymzm + l_xm * l_ym) * sizeof(float)));

    // Copy passed array to UM
    // Wanring: This can make managed epsx_f[i] accessable from CPU, but not from kernel, which will cause
    // CUDA_EXCEPTION_1/14 errors in the kernel.
    //epsx_f = epsx;
    //epsy_f = epsy;
    //epsz_f = epsz;
    hipMemcpy(epsx_f, epsx, (l_xmymzm + l_ym * l_zm) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(epsy_f, epsy, (l_xmymzm + l_xm * l_zm) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(epsz_f, epsz, (l_xmymzm + l_xm * l_ym) * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(devThreadsPerBlock, devThreadsPerBlock, devThreadsPerBlock);
    dim3 blocks((l_xm - 1)/devThreadsPerBlock + 1, (l_ym - 1) / devThreadsPerBlock + 1, (l_zm - 1) / devThreadsPerBlock + 1);
    feedepsintoam_kernel<<<blocks, threadsPerBlock>>>(l_xm, l_ym, l_zm, lepsx, lepsy, lepsz, epsx_f, epsy_f, epsz_f);
    cudaLaunchErrorCheck();

    // Next the salt term
    copy_vector_kernel<<<nblocks, blocksize>>>(l_iv, lp_iv, l_xmymzm);
    cudaLaunchErrorCheck();
    hipDeviceSynchronize();

    float lfactor = l_epsout * (l_h * l_pbkappa) * (l_h * l_pbkappa);

    // Finally we are ready to set up the A matrix
    // set up am/ad arrays at the finest level for all solvers
    // so only 1_xmymzm elements of leps* are initialized
    int j = 0;
    m = mg_index[j]; // m == 0 here
    int n = mg_index_ext[j];
    int lxmym = mg_size[j][0] * mg_size[j][1];
    // 1-D grid
    int lxmymzm = mg_size[j][0] * mg_size[j][1] * mg_size[j][2];
    dim3 h_threadsPerBlock (512);
    dim3 h_blocks((lxmymzm - 1) / 512 + 1);
    set_am_ad_kernel_head<<<h_blocks, h_threadsPerBlock>>>(lepsx + m, lepsy + m, lepsz + m, l_am1 + n + lxmym, l_am2 + n + lxmym, l_am3 + n + lxmym, lxmymzm);
    cudaLaunchErrorCheck();
    // 3-D grid
    set_am_ad_kernel_body<<<blocks, threadsPerBlock>>>(l_am1 + n + lxmym, l_am2 + n + lxmym, l_am3 + n + lxmym, l_ad + m, l_bz + m, l_iv + m, mg_size[j][0], mg_size[j][1], mg_size[j][2], lfactor, l_epsout);
    cudaLaunchErrorCheck();
    // 2-D grid
    dim3 t_threadsPerBlock(16, 16);
    dim3 t_blocks((max(mg_size[j][0], mg_size[j][1]) - 1) / 16 + 1, (max(mg_size[j][1], mg_size[j][2]) - 1) / 16 + 1);
    set_am_ad_kernel_tail<<<t_blocks, t_threadsPerBlock>>>(l_am1 + n + lxmym, l_am2 + n + lxmym, l_am3 + n + lxmym, mg_size[j][0], mg_size[j][1], mg_size[j][2]);
    cudaLaunchErrorCheck();

    hipDeviceSynchronize();

    if (*solvopt == 2) {
        for (j = 1; j < MG_NLEVEL; j++) {
            int l = mg_index[j-1];
            m = mg_index[j];
            n = mg_index_ext[j];

            lfactor *= 4;
            lxmym = mg_size[j][0] * mg_size[j][1];
            lxmymzm = mg_size[j][0] * mg_size[j][1] * mg_size[j][2];

            if (j < threshold) {
                // Resize
                dim3 blocks((mg_size[j][0] - 1)/devThreadsPerBlock + 1, (mg_size[j][1] - 1) / devThreadsPerBlock + 1, (mg_size[j][2] - 1) / devThreadsPerBlock + 1);
                restrict_eps_map_kernel<<<blocks, threadsPerBlock>>>(lepsx + l, lepsy + l, lepsz + l, mg_size[j - 1][0], mg_size[j - 1][1], mg_size[j - 1][2], lepsx + m, lepsy + m, lepsz+ m, mg_size[j][0], mg_size[j][1], mg_size[j][2]);
                cudaLaunchErrorCheck();
                restrict_v_kernel<<<blocks, threadsPerBlock>>>(64.0, l_iv + l, mg_size[j - 1][0], mg_size[j - 1][1], mg_size[j - 1][2], l_iv + m, mg_size[j][0], mg_size[j][1], mg_size[j][2]); //iv
                cudaLaunchErrorCheck();

                // 1-D grid
                dim3 h_blocks ((lxmymzm - 1) / 512 + 1);
                set_am_ad_kernel_head<<<h_blocks, h_threadsPerBlock>>>(lepsx + m, lepsy + m, lepsz + m, l_am1 + n + lxmym, l_am2 + n + lxmym, l_am3 + n + lxmym, lxmymzm);
                cudaLaunchErrorCheck();
                // 3-D grid
                set_am_ad_kernel_body<<<blocks, threadsPerBlock>>>(l_am1 + n + lxmym, l_am2 + n + lxmym, l_am3 + n + lxmym, l_ad + m, l_bz + m, l_iv + m, mg_size[j][0], mg_size[j][1], mg_size[j][2], lfactor, l_epsout);
                cudaLaunchErrorCheck();
                // 2-D grid
                dim3 t_blocks((max(mg_size[j][0], mg_size[j][1]) - 1) / 16 + 1, (max(mg_size[j][1], mg_size[j][2]) - 1) / 16 + 1);
                set_am_ad_kernel_tail<<<t_blocks, t_threadsPerBlock>>>(l_am1 + n + lxmym, l_am2 + n + lxmym, l_am3 + n + lxmym, mg_size[j][0], mg_size[j][1], mg_size[j][2]);
                cudaLaunchErrorCheck();

                hipDeviceSynchronize();

            } else {
                restrict_eps_map(lepsx + l, lepsy + l, lepsz + l, mg_size[j - 1][0], mg_size[j - 1][1], mg_size[j - 1][2], lepsx + m, lepsy + m, lepsz + m, mg_size[j][0], mg_size[j][1], mg_size[j][2]);
                restrict_v(64.0, l_iv + l, mg_size[j - 1][0], mg_size[j - 1][1], mg_size[j - 1][2], l_iv + m, mg_size[j][0], mg_size[j][1], mg_size[j][2]); //iv
                set_am_ad(lepsx + m, lepsy + m, lepsz + m, l_iv + m, l_am1 + n + lxmym, l_am2 + n + lxmym, l_am3 + n + lxmym, l_ad + m, l_bz + m, mg_size[j][0], mg_size[j][1], mg_size[j][2], lfactor, l_epsout);
            }
        }
    }
    hipFree(lp_bv);
    hipFree(lp_iv);
    hipFree(lp_xs);
    hipFree(lepsx);
    hipFree(lepsy);
    hipFree(lepsz);
    hipFree(epsx_f);
    hipFree(epsy_f);
    hipFree(epsz_f);
}

__global__
void init_vector_kernel(float *vec, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m) vec[i] = 0.0;
}

__global__
void copy_vector_kernel(float *vec, float *vec_f, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m) vec[i] = vec_f[i];
}

__global__
void inv_vector_kernel(float *vec, float *inv, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m) inv[i] = 1.0/vec[i];
}

__host__
void restrict_eps_map(float *epsxf, float *epsyf, float *epszf, int xn, int yn, int zn, float *epsxr, float *epsyr, float *epszr, int xnr, int ynr, int znr) {
    if (l_bcopt == 10) {
        printf("Not yet for PBC.\n");
        //Do nothing;
    } else {
        for(int k = 0; k < znr; k++) {
            int k2 = 2 * k + 1;
            for(int j = 0; j < ynr; j++) {
                int j2 = 2 * j + 1;
                for(int i = 0; i < xnr; i++) {
                    int i2 = 2 * i + 1;
                    int flatid = i + xnr * j + xnr * ynr * k;
                    // eps*r causes CUDA Exception 15 error. Solved
                    epsxr[flatid] = r_map_exp_x(epsxf, i2, j2, k2, xn, yn);
                    epsyr[flatid] = r_map_exp_y(epsyf, i2, j2, k2, xn, yn);
                    epszr[flatid] = r_map_exp_z(epszf, i2, j2, k2, xn, yn);
                }
            }
        }
    }
}

__host__ __device__
float r_map_exp_x(float *epsxmp, int i2, int j2, int k2, int xn, int yn) {
    float exp_x = hmav(epsxmp[f_id(i2  , j2  , k2  , xn, yn)], epsxmp[f_id(i2+1, j2  , k2  , xn, yn)]) / 4.0 +
                 (hmav(epsxmp[f_id(i2  , j2-1, k2  , xn, yn)], epsxmp[f_id(i2+1, j2-1, k2  , xn, yn)]) +
                  hmav(epsxmp[f_id(i2  , j2+1, k2  , xn, yn)], epsxmp[f_id(i2+1, j2+1, k2  , xn, yn)]) +
                  hmav(epsxmp[f_id(i2  , j2  , k2-1, xn, yn)], epsxmp[f_id(i2+1, j2  , k2-1, xn, yn)]) +
                  hmav(epsxmp[f_id(i2  , j2  , k2+1, xn, yn)], epsxmp[f_id(i2+1, j2  , k2+1, xn, yn)])) / 8.0 +
                 (hmav(epsxmp[f_id(i2  , j2-1, k2-1, xn, yn)], epsxmp[f_id(i2+1, j2-1, k2-1, xn, yn)]) +
                  hmav(epsxmp[f_id(i2  , j2+1, k2-1, xn, yn)], epsxmp[f_id(i2+1, j2+1, k2-1, xn, yn)]) +
                  hmav(epsxmp[f_id(i2  , j2-1, k2+1, xn, yn)], epsxmp[f_id(i2+1, j2-1, k2+1, xn, yn)]) +
                  hmav(epsxmp[f_id(i2  , j2+1, k2+1, xn, yn)], epsxmp[f_id(i2+1, j2+1, k2+1, xn, yn)])) / 16.0;

    // Debug
    //raise(SIGINT);
    return exp_x;
}

__host__ __device__
float r_map_exp_y(float *epsymp, int i2, int j2, int k2, int xn, int yn) {
    float exp_y = hmav(epsymp[f_id(i2  , j2  , k2  , xn, yn)], epsymp[f_id(i2  , j2+1, k2  , xn, yn)]) / 4.0 +
                 (hmav(epsymp[f_id(i2-1, j2  , k2  , xn, yn)], epsymp[f_id(i2-1, j2+1, k2  , xn, yn)]) +
                  hmav(epsymp[f_id(i2+1, j2  , k2  , xn, yn)], epsymp[f_id(i2+1, j2+1, k2  , xn, yn)]) +
                  hmav(epsymp[f_id(i2  , j2  , k2-1, xn, yn)], epsymp[f_id(i2  , j2+1, k2-1, xn, yn)]) +
                  hmav(epsymp[f_id(i2  , j2  , k2+1, xn, yn)], epsymp[f_id(i2  , j2+1, k2+1, xn, yn)])) / 8.0 +
                 (hmav(epsymp[f_id(i2-1, j2  , k2-1, xn, yn)], epsymp[f_id(i2-1, j2+1, k2-1, xn, yn)]) +
                  hmav(epsymp[f_id(i2+1, j2  , k2-1, xn, yn)], epsymp[f_id(i2+1, j2+1, k2-1, xn, yn)]) +
                  hmav(epsymp[f_id(i2-1, j2  , k2+1, xn, yn)], epsymp[f_id(i2-1, j2+1, k2+1, xn, yn)]) +
                  hmav(epsymp[f_id(i2+1, j2  , k2+1, xn, yn)], epsymp[f_id(i2+1, j2+1, k2+1, xn, yn)])) / 16.0;
    return exp_y;
}

__host__ __device__
float r_map_exp_z(float *epszmp, int i2, int j2, int k2, int xn, int yn) {
    float exp_z = hmav(epszmp[f_id(i2  , j2  , k2  , xn, yn)], epszmp[f_id(i2  , j2  , k2+1, xn, yn)]) / 4.0 +
                 (hmav(epszmp[f_id(i2  , j2-1, k2  , xn, yn)], epszmp[f_id(i2  , j2-1, k2+1, xn, yn)]) +
                  hmav(epszmp[f_id(i2  , j2+1, k2  , xn, yn)], epszmp[f_id(i2  , j2+1, k2+1, xn, yn)]) +
                  hmav(epszmp[f_id(i2-1, j2  , k2  , xn, yn)], epszmp[f_id(i2-1, j2  , k2+1, xn, yn)]) +
                  hmav(epszmp[f_id(i2+1, j2  , k2  , xn, yn)], epszmp[f_id(i2+1, j2  , k2+1, xn, yn)])) / 8.0 +
                 (hmav(epszmp[f_id(i2-1, j2-1, k2  , xn, yn)], epszmp[f_id(i2-1, j2-1, k2+1, xn, yn)]) +
                  hmav(epszmp[f_id(i2-1, j2+1, k2  , xn, yn)], epszmp[f_id(i2-1, j2+1, k2+1, xn, yn)]) +
                  hmav(epszmp[f_id(i2+1, j2-1, k2  , xn, yn)], epszmp[f_id(i2+1, j2-1, k2+1, xn, yn)]) +
                  hmav(epszmp[f_id(i2+1, j2+1, k2  , xn, yn)], epszmp[f_id(i2+1, j2+1, k2+1, xn, yn)])) / 16.0;
    return exp_z;
}

__host__ __device__
float hmav(float a, float b) {
    return 2.0 * a * b / (a + b);
}

__global__
void restrict_eps_map_kernel(float *epsxf, float *epsyf, float *epszf, int xn, int yn, int zn, float *epsxr, float *epsyr, float *epszr, int xnr, int ynr, int znr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    int i2 = 2 * i + 1;
    int j2 = 2 * j + 1;
    int k2 = 2 * k + 1;
    if (i < xnr  && j < ynr  && k < znr){
        int flatid = i + xnr * j + xnr * ynr * k;
        epsxr[flatid] = r_map_exp_x(epsxf, i2, j2, k2, xn, yn);
        epsyr[flatid] = r_map_exp_y(epsyf, i2, j2, k2, xn, yn);
        epszr[flatid] = r_map_exp_z(epszf, i2, j2, k2, xn, yn);
    }
}

// Assign on device
__global__
void feedepsintoam_kernel(int lxm, int lym, int lzm, float *am1, float *am2, float *am3, float *eps1, float *eps2, float *eps3) {
    // eps1 has (0:xm, ym, zm) dimension (0:ym/0:zm for eps2/eps3), which contains am1 (xm, ym, zm), so need to recheck the passed array index.
    // Edit: Solved, using 3-D mapping
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < lxm && j < lym && k < lzm) {
        int flatid = i + lxm * j + lxm * lym * k;
        // Warning of bug, x/y dimensions are incorrect, that's why only lepsz is good
        //int flatid_x_plus1 = (i + 1) + lxm * j + lxm * lym * k;
        //int flatid_y_plus1 = i + lxm * (j + 1) + lxm * lym * k;
        int flatid_x_plus1 = (i + 1) + (lxm + 1) * j + (lxm + 1) * lym * k;
        int flatid_y_plus1 = i + lxm * (j + 1) + lxm * (lym + 1) * k;
        int flatid_z_plus1 = i + lxm * j + lxm * lym * (k + 1);

        // eps* causes CUDA Exception 14 error. Solved
        am1[flatid] = eps1[flatid_x_plus1];
        am2[flatid] = eps2[flatid_y_plus1];
        am3[flatid] = eps3[flatid_z_plus1];
    }
}

__host__
void set_am_ad(float *epsx, float *epsy, float *epsz, float *iv, float *lam1, float *lam2, float *lam3, float *lad, float *lbz, int xn, int yn, int zn, float lfactor, float epsout) {
    for (int i = 0; i < xn * yn * zn; i++) {
        lam1[i] = epsx[i];
        lam2[i] = epsy[i];
        lam3[i] = epsz[i];
    }

    for (int k = 0; k < zn; k++) {
        for (int j = 0; j < yn; j++) {
            for (int i = 0; i < xn; i++) {
                int flatid = i + xn * j + xn * yn * k;
                int flatid_x_minus1 = (i - 1) + xn * j + xn * yn * k;
                int flatid_y_minus1 = i + xn * (j - 1) + xn * yn * k;
                int flatid_z_minus1 = i + xn * j + xn * yn * (k - 1);

                lad[flatid] = lam1[flatid] + lam2[flatid] + lam3[flatid];
                if (i == 0) lad[flatid] += epsout; else lad[flatid] += lam1[flatid_x_minus1];
                if (j == 0) lad[flatid] += epsout; else lad[flatid] += lam2[flatid_y_minus1];
                if (k == 0) lad[flatid] += epsout; else lad[flatid] += lam3[flatid_z_minus1];

                lbz[flatid] = lfactor * iv[flatid];
                lad[flatid] += lbz[flatid];
            }
        }
    }

    if (l_bcopt != 10) {
        for (int k = 0; k < zn; k++) {
            for (int j = 0; j < yn; j++) {
                for (int i = 0; i < xn; i++) {
                    int flatid = i + xn * j + xn * yn * k;
                    if (i == xn - 1) lam1[flatid] = 0;
                    if (j == yn - 1) lam2[flatid] = 0;
                    if (k == zn - 1) lam3[flatid] = 0;
                }
            }
        }
    }
}

__global__
void set_am_ad_kernel_head(float *epsx, float *epsy, float *epsz, float *lam1, float *lam2, float *lam3, int xnynzn) {
    // 1-D, xnynzn
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < xnynzn) {
        lam1[i] = epsx[i];
        lam2[i] = epsy[i];
        lam3[i] = epsz[i];
    }
}

__global__
void set_am_ad_kernel_body(float *lam1, float *lam2, float *lam3, float *lad, float *lbz, float *iv, int xn, int yn, int zn, float lfactor, float epsout) {
    // 3-D, (xn, yn, zn)
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < xn && j < yn && k < zn) {
        int flatid = i + xn * j + xn * yn * k;
        int flatid_x_minus1 = (i - 1) + xn * j + xn * yn * k;
        int flatid_y_minus1 = i + xn * (j - 1) + xn * yn * k;
        int flatid_z_minus1 = i + xn * j + xn * yn * (k - 1);

        lad[flatid] = lam1[flatid] + lam2[flatid] + lam3[flatid];
        if (i == 0) lad[flatid] += epsout; else lad[flatid] += lam1[flatid_x_minus1];
        if (j == 0) lad[flatid] += epsout; else lad[flatid] += lam2[flatid_y_minus1];
        if (k == 0) lad[flatid] += epsout; else lad[flatid] += lam3[flatid_z_minus1];

        lbz[flatid] = lfactor * iv[flatid];
        lad[flatid] += lbz[flatid];
    }
}

__global__
void set_am_ad_kernel_tail(float *lam1, float *lam2, float *lam3, int xn, int yn, int zn) {
    // 2-D, (i, j) -> (max_xy, max_yz)
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < yn && j < zn) {
        lam1[xn - 1 + xn * i + xn * yn * j] = 0.0;
    }

    if (i < xn && j < zn) {
        lam2[i + xn * (yn - 1) + xn * yn * j] = 0.0;
    }

    if (i < xn && j < yn) {
        lam2[i + xn * j + xn * yn * (zn - 1)] = 0.0;
    }
}

__host__
void restrict_v(float divider, float *bvf, int nx, int ny, int nz, float *bvr, int nxr, int nyr, int nzr) {
    for(int k = 0; k < nzr; k++) {
        int k2 = 2 * k + 1;
        for(int j = 0; j < nyr; j++) {
            int j2 = 2 * j + 1;
            for(int i = 0; i < nxr; i++) {
                int i2 = 2 * i + 1;
                int flatid = i + nxr * j + nxr * nyr * k;
                bvr[flatid] = ( bvf[f_id(i2-1, j2-1, k2-1, nx, ny)] + 2 * bvf[f_id(i2, j2-1, k2-1, nx, ny)] + bvf[f_id(i2+1, j2-1, k2-1, nx, ny)] ) +
                          2 * ( bvf[f_id(i2-1, j2  , k2-1, nx, ny)] + 2 * bvf[f_id(i2, j2  , k2-1, nx, ny)] + bvf[f_id(i2+1, j2  , k2-1, nx, ny)] ) +
                              ( bvf[f_id(i2-1, j2+1, k2-1, nx, ny)] + 2 * bvf[f_id(i2, j2+1, k2-1, nx, ny)] + bvf[f_id(i2+1, j2+1, k2-1, nx, ny)] ) +
                          2 * ( bvf[f_id(i2-1, j2-1, k2  , nx, ny)] + 2 * bvf[f_id(i2, j2-1, k2  , nx, ny)] + bvf[f_id(i2+1, j2-1, k2  , nx, ny)] ) +
                          4 * ( bvf[f_id(i2-1, j2  , k2  , nx, ny)] + 2 * bvf[f_id(i2, j2  , k2  , nx, ny)] + bvf[f_id(i2+1, j2  , k2  , nx, ny)] ) +
                          2 * ( bvf[f_id(i2-1, j2+1, k2  , nx, ny)] + 2 * bvf[f_id(i2, j2+1, k2  , nx, ny)] + bvf[f_id(i2+1, j2+1, k2  , nx, ny)] ) +
                              ( bvf[f_id(i2-1, j2-1, k2+1, nx, ny)] + 2 * bvf[f_id(i2, j2-1, k2+1, nx, ny)] + bvf[f_id(i2+1, j2-1, k2+1, nx, ny)] ) +
                          2 * ( bvf[f_id(i2-1, j2  , k2+1, nx, ny)] + 2 * bvf[f_id(i2, j2  , k2+1, nx, ny)] + bvf[f_id(i2+1, j2  , k2+1, nx, ny)] ) +
                              ( bvf[f_id(i2-1, j2+1, k2+1, nx, ny)] + 2 * bvf[f_id(i2, j2+1, k2+1, nx, ny)] + bvf[f_id(i2+1, j2+1, k2+1, nx, ny)] );
                bvr[flatid] /= divider;
            }
        }
    }
}

__global__
void restrict_v_kernel(float divider, float *bvf, int nx, int ny, int nz, float *bvr, int nxr, int nyr, int nzr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    int i2 = 2 * i + 1;
    int j2 = 2 * j + 1;
    int k2 = 2 * k + 1;
    if (i < nxr  && j < nyr  && k < nzr){
        int b_id = i + nxr * j + nxr * nyr * k;

        bvr[b_id] = ( bvf[f_id(i2-1, j2-1, k2-1, nx, ny)] + 2 * bvf[f_id(i2, j2-1, k2-1, nx, ny)] + bvf[f_id(i2+1, j2-1, k2-1, nx, ny)] ) +
                2 * ( bvf[f_id(i2-1, j2  , k2-1, nx, ny)] + 2 * bvf[f_id(i2, j2  , k2-1, nx, ny)] + bvf[f_id(i2+1, j2  , k2-1, nx, ny)] ) +
                    ( bvf[f_id(i2-1, j2+1, k2-1, nx, ny)] + 2 * bvf[f_id(i2, j2+1, k2-1, nx, ny)] + bvf[f_id(i2+1, j2+1, k2-1, nx, ny)] ) +
                2 * ( bvf[f_id(i2-1, j2-1, k2  , nx, ny)] + 2 * bvf[f_id(i2, j2-1, k2  , nx, ny)] + bvf[f_id(i2+1, j2-1, k2  , nx, ny)] ) +
                4 * ( bvf[f_id(i2-1, j2  , k2  , nx, ny)] + 2 * bvf[f_id(i2, j2  , k2  , nx, ny)] + bvf[f_id(i2+1, j2  , k2  , nx, ny)] ) +
                2 * ( bvf[f_id(i2-1, j2+1, k2  , nx, ny)] + 2 * bvf[f_id(i2, j2+1, k2  , nx, ny)] + bvf[f_id(i2+1, j2+1, k2  , nx, ny)] ) +
                    ( bvf[f_id(i2-1, j2-1, k2+1, nx, ny)] + 2 * bvf[f_id(i2, j2-1, k2+1, nx, ny)] + bvf[f_id(i2+1, j2-1, k2+1, nx, ny)] ) +
                2 * ( bvf[f_id(i2-1, j2  , k2+1, nx, ny)] + 2 * bvf[f_id(i2, j2  , k2+1, nx, ny)] + bvf[f_id(i2+1, j2  , k2+1, nx, ny)] ) +
                    ( bvf[f_id(i2-1, j2+1, k2+1, nx, ny)] + 2 * bvf[f_id(i2, j2+1, k2+1, nx, ny)] + bvf[f_id(i2+1, j2+1, k2+1, nx, ny)] );
        bvr[b_id] /= divider;
    }
}

__host__ __device__
int f_id(int i, int j, int k, int nx, int ny) {
    return i + nx * j + nx * ny * k;
}

__host__
void restrict_cuda(int level) {
    float div = 16.0; //restrict_bv

    int nx = mg_size[level][0];
    int ny = mg_size[level][1];
    int nz = mg_size[level][2];
    int nxr = mg_size[level + 1][0];
    int nyr = mg_size[level + 1][1];
    int nzr = mg_size[level + 1][2];

    dim3 threadsPerBlock(devThreadsPerBlock, devThreadsPerBlock, devThreadsPerBlock);
    dim3 blocks((nx - 1)/devThreadsPerBlock + 1, (ny - 1) / devThreadsPerBlock + 1, (nz - 1) / devThreadsPerBlock +1);
    restrict_v_kernel<<<blocks, threadsPerBlock>>>(div, l_rv + mg_index[level], nx, ny, nz, l_bv + mg_index[level + 1], nxr, nyr, nzr);
    cudaLaunchErrorCheck();
    hipDeviceSynchronize();
}

__host__
void interpolate(int level) {
    // Derive all vectors
    int xn = mg_size[level + 1][0];
    int yn = mg_size[level + 1][1];
    int zn = mg_size[level + 1][2];
    int xni = mg_size[level][0];
    int yni = mg_size[level][1];
    int zni = mg_size[level][2];
    int xniyni = xni * yni;
    int xniynizni = xni * yni * zni;

    int p1 = mg_x_idx[level + 1] + mg_size[level + 1][0] * mg_size[level + 1][1];
    int p2 = mg_x_idx[level] + mg_size[level][0] * mg_size[level][1];

    float *v = &l_xv[p1];
    float *vi = &l_xv[p2];
    float *lam1 = &l_am1[mg_index_ext[level]]; // 1-xniyni:~
    float *lam2 = &l_am2[mg_index_ext[level]];
    float *lam3 = &l_am3[mg_index_ext[level]];
    float *lbz = &l_bz[mg_index[level]];
    float epsout = l_epsout;

    if (xn * 2 + 1 != xni || yn * 2 + 1 != yni || zn * 2 + 1 != zni) {
        printf("Interpolation failed because of incorrect dimension (interpolate_host)\n");
        printf("xn %d, yn %d, zn %d\n", xn, yn, zn);
        printf("xni %d, yni %d, zni %d\n", xni, yni, zni);
        exit(2);
    }

    //raise(SIGINT);

    /*
    for(int k = 0; k < zni; k++) {
        for(int j = 0; j < yni; j++) {
            for(int i = 0; i < xni; i++) {
                int flatid_c = i + xni * j + xni * yni * k; // Coarse grid
                // Later handel lam value of epsout at points on surfaces (i-1/j-1/k-1). TBD
                // Use 3-D index to manually add epsout value with outter surface
                // Check first what lam values were used inside each function
                // The original code wasted too much with 1-xniyni:0 range. Need to use 3-D surface to computate the
                // contribution at each point.
                // For now, temporarily use original index, add any offset needed here to fetch the correct position
                // Since am1/2/3 is used as 1-D array only in ipl_comp* functions, add offset there will do the trick.
                // Edit: the index of am* is linear to l, so let l += offset, and finially flatid_f += offset.
                // Edit: Don't add to flatid_f, which will cause error to vi; add to am* index in ipl_com* func only.

                //if (i == 0) lam1[flatid_c] += ;

                // Surfaces around vertices
                // Mapping: Thread block --> Coarse grid block --> Fine grid block
                if (i == xni) lam1[flatid_c] = epsout;
                if (j == yni) lam2[flatid_c] = epsout;
                if (k == zni) lam3[flatid_c] = epsout;

            }
        }
    }*/

    // Initialize to epsout, should be before ipl_chain function calls
    // 1-xniyni:0, outside 3-D loop
    for ( int i = 0; i < xniyni; i++) {
        lam1[i] = epsout;
        lam2[i] = epsout;
        lam3[i] = epsout;
    }

    // Three surfaces
    // Need adding index offset. TBF. Fixed
    for (int k = 0; k < zni; k++) {
        for (int j = 0; j < yni; j++) {
            // [(xni - 1) + j * xni + k * xniyni] + xniyni
            lam1[-1 + (j + 1) * xni + (k + 1) * xniyni] = epsout;
        }
    }
    for (int k = 0; k < zni; k++) {
        for (int i = 0; i < xni; i++) {
            // [i + (yni - 1) * xni + k * xniyni] + xniyni
            lam2[i - xni + (k + 2) * xniyni] = epsout;
        }
    }
    for (int j = 0; j < yni; j++) {
        for (int i = 0; i < xni; i++) {
            lam3[i + j * xni + xniynizni] = epsout;
        }
    }

    for(int k = 0; k < zn; k++) {
        for(int j = 0; j < yn; j++) {
            for(int i = 0; i < xn; i++) {
                // Caution with the indexes! Starting from 0
                int flatid_c = i + xn * j + xn * yn * k; // Coarse grid
                int flatid_f = (2 * i + 1) + (2 * j + 1) * xni + (2 * k + 1) * xniyni; // Fine grid

                vi[flatid_f] += v[flatid_c];

                // offset for offsetting the index
                ipl_chain_h(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam1,      -1, lam2, xni, lam3, xniyni, xn, yn, zn);
                ipl_chain_h(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam1,      +1, lam2, xni, lam3, xniyni, xn, yn, zn);
                ipl_chain_h(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam2,    -xni, lam1,   1, lam3, xniyni, xn, yn, zn);
                ipl_chain_h(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam2,    +xni, lam1,   1, lam3, xniyni, xn, yn, zn);
                ipl_chain_h(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam3, -xniyni, lam2, xni, lam1,      1, xn, yn, zn);
                ipl_chain_h(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam3, +xniyni, lam2, xni, lam1,      1, xn, yn, zn);
            }
        }
    }

    // 1-xniyni:0, outside 3-D loop
    for ( int i = 0; i < xniyni; i++) {
        lam1[i] = 0.0;
        lam2[i] = 0.0;
        lam3[i] = 0.0;
    }

    // Three surfaces
    for (int k = 0; k < zni; k++) {
        for (int j = 0; j < yni; j++) {
            lam1[-1 + (j + 1) * xni + (k + 1) * xniyni] = 0.0;
        }
    }
    for (int k = 0; k < zni; k++) {
        for (int i = 0; i < xni; i++) {
            lam2[i - xni + (k + 2) * xniyni] = 0.0;
        }
    }
    for (int j = 0; j < yni; j++) {
        for (int i = 0; i < xni; i++) {
            lam3[i + j * xni + xniynizni] = 0.0;
        }
    }

    // A better way to handel and fix all index issues, using index 1 ~ xnynzn, not
    // 1-nxny:nxnynz+nxny.
    // i.e. grid[level]->xs[0:lxlylz-1] v.s. (arrays + index dereference),
    // but at the expense of struct indrection overhead. Figure out to what degree.
    // May define a general Grid structure with all vector pointers inside, then create an array
    // Grid *grid[MG_NLEVEL],
    // then initialize all matrix vectors via grid[level]->xs = (float *) malloc();
}

__global__
void interpolate_kernel_head(int xni, int yni, int zni, float *lam1, float *lam2, float *lam3, float epsout) {
    int xniyni = xni * yni;
    // 2-D, [i, j] -> [max(xni, yni), max(yni, zni)]
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Padding offset <xniyni> included
    if (i < yni && j < zni) {
        lam1[-1 + (i + 1) * xni + (j + 1) * xniyni] = epsout;
    }

    if (i < xni && j < zni) {
        lam2[i - xni + (j + 2) * xniyni] = epsout;
    }

    if (i < xni && j < yni) {
        lam3[i + j * xni + xni * yni * zni] = epsout;
        // 0:xniyni-1
        lam1[i + j * xni] = epsout;
        lam2[i + j * xni] = epsout;
        lam3[i + j * xni] = epsout;
    }
}

__global__
void interpolate_kernel_body(float *v, int xn, int yn, int zn, float *vi, int xni, int yni, int zni, float *lam1, float *lam2, float *lam3, float *lbz) {
    int xniyni = xni * yni;
    int xniynizni = xni * yni * zni;

    // 3-D, (xn, yn, zn)
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < xn  && j < yn  && k < zn){
        int flatid_c = i + xn * j + xn * yn * k; // Coarse grid
        int flatid_f = (2 * i + 1) + (2 * j + 1) * xni + (2 * k + 1) * xniyni; // Fine grid

        vi[flatid_f] += v[flatid_c];

        // offset for offsetting the index
        ipl_chain_d(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam1,      -1, lam2, xni, lam3, xniyni, xn, yn, zn);
        ipl_chain_d(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam1,      +1, lam2, xni, lam3, xniyni, xn, yn, zn);
        ipl_chain_d(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam2,    -xni, lam1,   1, lam3, xniyni, xn, yn, zn);
        ipl_chain_d(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam2,    +xni, lam1,   1, lam3, xniyni, xn, yn, zn);
        ipl_chain_d(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam3, -xniyni, lam2, xni, lam1,      1, xn, yn, zn);
        ipl_chain_d(vi, xniyni, xniynizni, flatid_f, *(v + flatid_c), lbz, lam3, +xniyni, lam2, xni, lam1,      1, xn, yn, zn);
    }

}

// Refine the four types point value assignment using directly thread block, layer by layer,
// pass single array value distributed on single thread. TBD
__host__
void ipl_chain_h(float *vi, int xnyn, int xnynzn, int l, float v, float *lbz, float *am_1, int shift_1, float *am_2, int shift_2, float *am_3, int shift_3, int xn, int yn, int zn) {
    // Here v is value of coarse grid; vi is array of fine grid
    float v1 = ipl_comp1(v, l, lbz, am_1, xnyn, xnynzn, shift_1);
    int l1 = l + shift_1;
    vi[l1] += v1;
    ipl_chain2_h(vi, xnyn, xnynzn, l1, v1, lbz, am_1, shift_1, am_2, -shift_2, am_3, shift_3, xn, yn, zn);
    ipl_chain2_h(vi, xnyn, xnynzn, l1, v1, lbz, am_1, shift_1, am_2,  shift_2, am_3, shift_3, xn, yn, zn);
    ipl_chain2_h(vi, xnyn, xnynzn, l1, v1, lbz, am_1, shift_1, am_3, -shift_3, am_2, shift_2, xn, yn, zn);
    ipl_chain2_h(vi, xnyn, xnynzn, l1, v1, lbz, am_1, shift_1, am_3,  shift_3, am_2, shift_2, xn, yn, zn);
}

__device__
void ipl_chain_d(float *vi, int xnyn, int xnynzn, int l, float v, float *lbz, float *am_1, int shift_1, float *am_2, int shift_2, float *am_3, int shift_3, int xn, int yn, int zn) {
    // Here v is value of coarse grid; vi is array of fine grid
    float v1 = ipl_comp1(v, l, lbz, am_1, xnyn, xnynzn, shift_1);
    int l1 = l + shift_1;
    atomicAdd(&vi[l1], v1);
    ipl_chain2_d(vi, xnyn, xnynzn, l1, v1, lbz, am_1, shift_1, am_2, -shift_2, am_3, shift_3, xn, yn, zn);
    ipl_chain2_d(vi, xnyn, xnynzn, l1, v1, lbz, am_1, shift_1, am_2,  shift_2, am_3, shift_3, xn, yn, zn);
    ipl_chain2_d(vi, xnyn, xnynzn, l1, v1, lbz, am_1, shift_1, am_3, -shift_3, am_2, shift_2, xn, yn, zn);
    ipl_chain2_d(vi, xnyn, xnynzn, l1, v1, lbz, am_1, shift_1, am_3,  shift_3, am_2, shift_2, xn, yn, zn);
}

__host__
void ipl_chain2_h(float *vi, int xnyn, int xnynzn, int l1, float v1, float *lbz, float *am_1, int shift_1, float *am_2, int shift_2, float *am_3, int shift_3, int xn, int yn, int zn) {
    float v2 = ipl_comp2(v1, l1, lbz, am_1, am_2, xnyn, xnynzn, shift_1, shift_2);
    int l2 = l1 + shift_2;
    vi[l2] += v2;
    vi[l2 - shift_3] += ipl_comp3(v2, l2, lbz, am_1, am_2, am_3, xnyn, xnynzn, shift_1, shift_2, -shift_3);
    vi[l2 + shift_3] += ipl_comp3(v2, l2, lbz, am_1, am_2, am_3, xnyn, xnynzn, shift_1, shift_2, +shift_3);
}

__device__
void ipl_chain2_d(float *vi, int xnyn, int xnynzn, int l1, float v1, float *lbz, float *am_1, int shift_1, float *am_2, int shift_2, float *am_3, int shift_3, int xn, int yn, int zn) {
    float v2 = ipl_comp2(v1, l1, lbz, am_1, am_2, xnyn, xnynzn, shift_1, shift_2);
    int l2 = l1 + shift_2;
    atomicAdd(&vi[l2], v2);
    float v3 = ipl_comp3(v2, l2, lbz, am_1, am_2, am_3, xnyn, xnynzn, shift_1, shift_2, -shift_3);
    atomicAdd(&vi[l2 - shift_3], v3);
    float v4 = ipl_comp3(v2, l2, lbz, am_1, am_2, am_3, xnyn, xnynzn, shift_1, shift_2, +shift_3);
    atomicAdd(&vi[l2 + shift_3], v4);
}

__host__ __device__
float ipl_comp1(float v, int l, float *lbz, float *am_1, int xnyn, int xnynzn, int shift_1) {
    float ipl_comp1_v;
    // For offsetting
    int bz_l = l;
    l += xnyn;

    if (shift_1 < 0)
        ipl_comp1_v = v * am_1[l + shift_1] / ( lbz[bz_l + shift_1] + am_1[l + 2 * shift_1] + am_1[l + shift_1] );
    else
        ipl_comp1_v = v * am_1[l] / ( lbz[bz_l + shift_1] + am_1[l] + am_1[l + shift_1] );
    return ipl_comp1_v;
}

__host__ __device__
float ipl_comp2(float v, int l, float *lbz, float *am_1, float *am_2, int xnyn, int xnynzn, int shift_1, int shift_2) {
    // For offsetting
    int bz_l = l;
    l += xnyn;

    float lad = am_1[l + shift_2] + am_1[l + shift_2 - abs(shift_1)] + lbz[bz_l + shift_2];
    float ipl_comp2_v;

    if (shift_2 < 0)
        ipl_comp2_v = v * am_2[l + shift_2] / ( am_2[l + 2 * shift_2] + am_2[l + shift_2] + lad);
    else
        ipl_comp2_v = v * am_2[l] / ( am_2[l] + am_2[l + shift_2] + lad );
    return ipl_comp2_v;
}

__host__ __device__
float ipl_comp3(float v, int l, float *lbz, float *am_1, float *am_2, float *am_3, int xnyn, int xnynzn, int shift_1, int shift_2, int shift_3) {
    // For offsetting
    int bz_l = l;
    l += xnyn;

    float lad = am_1[l + shift_3] + am_1[l + shift_3 - abs(shift_1)] + am_2[l + shift_3] + am_2[l + shift_3 - abs(shift_2)] + lbz[bz_l + shift_3];
    float ipl_comp3_v;

    if (shift_3 < 0)
        ipl_comp3_v = v * am_3[l + shift_3] / ( am_3[l + 2 * shift_3] + am_3[l + shift_3] + lad);
    else
        ipl_comp3_v = v * am_3[l] / ( am_3[l] + am_3[l + shift_3] + lad );
    return ipl_comp3_v;
}

__host__
void interpolate_cuda(int level) {
    int xn = mg_size[level + 1][0];
    int yn = mg_size[level + 1][1];
    int zn = mg_size[level + 1][2];
    int xni = mg_size[level][0];
    int yni = mg_size[level][1];
    int zni = mg_size[level][2];

    int p1 = mg_x_idx[level + 1] + mg_size[level + 1][0] * mg_size[level + 1][1];
    int p2 = mg_x_idx[level] + mg_size[level][0] * mg_size[level][1];
    int p3 = mg_index_ext[level];
    int p4 = mg_index[level];

    if (xn * 2 + 1 != xni || yn * 2 + 1 != yni || zn * 2 + 1 != zni) {
        printf("Interpolation failed because of incorrect dimension (interpolate_cuda)\n");
        printf("xn %d, yn %d, zn %d\n", xn, yn, zn);
        printf("xni %d, yni %d, zni %d\n", xni, yni, zni);
        exit(2);
    }

    /*
     * warning: Cuda API error detected: hipLaunchByPtr returned (0x7).
     *
     * From /usr/local/cuda-7.5/include/hip/driver_types.h:
     *      hipErrorLaunchOutOfResources         =      7,
     *      This indicates that a launch did not occur because it did not have
     *      appropriate resources. Although this error is similar to
     *      ::hipErrorInvalidConfiguration, this error usually indicates that the
     *      user has attempted to pass too many arguments to the device kernel, or the
     *      kernel launch specifies too many threads for the kernel's register count.
     *
     * Fixed temporarily. Use nvcc --maxrregcount=128
     * In release code this should be specified inside code, best using launch bounds:
     * http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#launch-bounds
     */

    // 2-D grid
    dim3 h_threadsPerBlock(16, 16);
    dim3 h_blocks((max(xni, yni) - 1) / 16 + 1, (max(yni, zni) - 1) / 16 + 1);
    interpolate_kernel_head<<<h_blocks, h_threadsPerBlock>>>(xni, yni, zni, l_am1 + p3, l_am2 + p3, l_am3 + p3, l_epsout);
    cudaLaunchErrorCheck();

    // Using (8, 8, 4) grid solved the 'CUDA launch failed: too many resources requested' issue.
    // Optimize the register use, block, grid size later. TBD
    dim3 threadsPerBlock(devThreadsPerBlock, devThreadsPerBlock, devThreadsPerBlock/2);
    dim3 blocks((xn - 1) / devThreadsPerBlock + 1, (yn - 1) / devThreadsPerBlock + 1, 2 * (zn - 1) / devThreadsPerBlock + 1);
    interpolate_kernel_body<<<blocks, threadsPerBlock>>>(l_xv + p1, xn, yn, zn, l_xv + p2, xni, yni, zni, l_am1 + p3, l_am2 + p3, l_am3 + p3, l_bz + p4);
    cudaLaunchErrorCheck();

    interpolate_kernel_head<<<h_blocks, h_threadsPerBlock>>>(xni, yni, zni, l_am1 + p3, l_am2 + p3, l_am3 + p3, 0.0);
    cudaLaunchErrorCheck();
    hipDeviceSynchronize();
}

__host__
void relax(int level, int ncyc) {
    int nx = mg_size[level][0];
    int ny = mg_size[level][1];
    int nz = mg_size[level][2];
    int nxny = nx * ny;
    int nxnynz = nxny * nz;
    float *xs = l_xv + mg_x_idx[level];
    float *lam1 = l_am1 + mg_index_ext[level];
    float *lam2 = l_am2 + mg_index_ext[level];
    float *lam3 = l_am3 + mg_index_ext[level];
    float *lzv = l_zv + mg_index[level];
    float *lad = l_ad + mg_index[level];
    float *lbv = l_bv + mg_index[level];
    float *lrv = l_rv + mg_index[level];
    float accept = l_accept;

    float onorm = mg_onorm[level];

    int itn_checknorm;
    float wsor;//, wsor1;
    float linorm = 0.0;
    float lnorm;
    int itmax = 1000; // Should move to upper layer

    if (ncyc > 0) {
        itn_checknorm = ncyc;
        wsor = 1.0; //1.0; Debug: 1.0 is Gauss-Seidel
    } else {
        itn_checknorm = 10;
        wsor = 1.9; // 1.0; Debug of omega - use GS to solve on coarsest level
    }

    for (int i = 0; i < nxnynz; i++) {
        linorm += abs(lbv[i]);
        lzv[i] = 1.0 / lad[i];
    }

    bool converged = false;
    int litn = 0;
    while (!converged) {
        int i;
        for (i = nxny;  i < nxnynz+nxny; i++) {
            xs[i] -= wsor * (xs[i] - (lam1[i - 1   ] * xs[i - 1   ] + lam1[i         ] * xs[i + 1   ] +
                                      lam2[i - nx  ] * xs[i - nx  ] + lam2[i         ] * xs[i + nx  ] +
                                      lam3[i - nxny] * xs[i - nxny] + lam3[i         ] * xs[i + nxny] + lbv[i - nxny]) * lzv[i - nxny]);
        }

        litn++;

        // Check convergence
        if (litn % itn_checknorm == 0) {
            // residue
            for (int i = nxny; i < nxnynz+nxny; i++) {
                lrv[i - nxny] = lam1[i - 1   ] * xs[i - 1   ] + lam1[i       ] * xs[i + 1   ] +
                                lam2[i - nx  ] * xs[i - nx  ] + lam2[i       ] * xs[i + nx  ] +
                                lam3[i - nxny] * xs[i - nxny] + lam3[i       ] * xs[i + nxny] + lbv[i - nxny] - lad[i - nxny] * xs[i];
                }

            // norm
            lnorm = 0.0;
            for (int i = 0; i < nxnynz; i++) {
                lnorm += abs(lrv[i]);
            }

            if (litn >= itmax || (ncyc > 0 && (litn >= ncyc && lnorm < onorm)) || lnorm <= accept * linorm) {
                converged = true;
                if (ncyc > 0 && litn >= ncyc && lnorm > onorm) {
                    printf("PB_MG FAILED: ncyc %d\t, itn %d\t, norm %e\t, onorm %e\n", ncyc, litn, lnorm, onorm);
                    exit(2);
                }

                if (ncyc > 0) mg_onorm[level] = lnorm; // Update global array
                if (litn >= itmax) printf("PB_MG WARNING: SOR maxitn exceeded (relax_host)!\n");
            }

        }
    } // while
}

__host__
void relax_cuda(int level, int ncyc) {
    int threadsPerBlock = 512; // This shouldn't be final, RL

    int nx = mg_size[level][0];
    int ny = mg_size[level][1];
    int nz = mg_size[level][2];
    int nxny = nx * ny;
    int nxnynz = nxny * nz;

    float *xs = l_xv + mg_x_idx[level];
    float *lam1 = l_am1 + mg_index_ext[level];
    float *lam2 = l_am2 + mg_index_ext[level];
    float *lam3 = l_am3 + mg_index_ext[level];
    float *lzv = l_zv + mg_index[level];
    float *lad = l_ad + mg_index[level];
    float *lbv = l_bv + mg_index[level];
    float *lrv = l_rv + mg_index[level];
    float accept = l_accept;
    float onorm = mg_onorm[level];

    int itn_checknorm = 10;
    float wsor;//, wsor1;
    int itmax = 1000;

    if (ncyc > 0) {
        itn_checknorm = ncyc;
        wsor = 1.0;// 1.0 Test
    } else {
        itn_checknorm = 10;
        wsor = 1.9; // 1.9 Test
    }

    ncyc = 10; // This shouldn't be final, RL

    // inverse AD for fast processing later
    int blocks = (nxnynz - 1) / threadsPerBlock + 1;
    inv_vector_kernel<<<blocks, threadsPerBlock>>>(lad, lzv, nxnynz);
    cudaLaunchErrorCheck();

    // initial norm. Create cuBlAS context
    float linorm = 0.0;
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    cublasErrorCheck(status);
    status = hipblasSasum(handle, nxnynz, lbv, 1, &linorm);
    cublasErrorCheck(status);

    bool converged = false;
    int litn = 0;
    int sblocks = ((int)(nxnynz / 2) + (nxnynz & 1) - 1) / threadsPerBlock + 1;
    while (!converged) {
        // non-periodic
        solver_red_kernel<<<sblocks, threadsPerBlock>>>(xs, lam1, lam2, lam3, lzv, lbv, wsor, nx, ny, nz);
        cudaLaunchErrorCheck();
        solver_black_kernel<<<sblocks, threadsPerBlock>>>(xs, lam1, lam2, lam3, lzv, lbv, wsor, nx, ny, nz);
        cudaLaunchErrorCheck();
        //hipDeviceSynchronize(); // Warning of bug: CUDA_EXCEPTION_15

        litn++;

        // Check convergence
        if (litn % itn_checknorm == 0) {
            residue_kernel<<<blocks, threadsPerBlock>>>(xs, lam1, lam2, lam3, lad, lbv, nx, ny, nz, lrv);
            cudaLaunchErrorCheck();

            // Create cuBlAS context
            float lnorm = 0.0;
            status = hipblasSasum(handle, nxnynz, lrv, 1, &lnorm);
            cublasErrorCheck(status);

            if (litn >= itmax || (ncyc > 0 && (litn >= ncyc && lnorm < onorm)) || lnorm <= accept * linorm) {
                converged = true;
                if (ncyc > 0 && litn >= ncyc && lnorm > onorm) {
                    printf("PB_MG FAILED: ncyc %d\t, itn %d\t, norm %e\t, onorm %e\n", ncyc, litn, lnorm, onorm);
                    exit(2);
                }

                if (ncyc > 0) mg_onorm[level] = lnorm; // Update global array
                if (litn >= itmax) printf("PB_MG WARNING: SOR maxitn exceeded (relax_kernel)!\n");
            }
        } // if
    } // while

    // Destroy context
    hipblasDestroy(handle);
}

__global__
void solver_black_kernel(float *phi, float *epsi, float *epsj, float *epsk,float *repsc, float *rho, float wsor, int xm, int ym, int zm) {

    int xmym = xm * ym;
    int i = 2 * (blockIdx.x * blockDim.x + threadIdx.x) + xmym;

    if (i < xm * ym * zm + xmym) {
        phi[i] -= wsor * (phi[i] - (epsi[i - 1   ] * phi[i - 1   ] + epsi[i      ] * phi[i + 1   ] +
                                    epsj[i - xm  ] * phi[i - xm  ] + epsj[i      ] * phi[i + xm  ] +
                                    epsk[i - xmym] * phi[i - xmym] + epsk[i      ] * phi[i + xmym] + rho[i - xmym]) * repsc[i - xmym]);
    }

}

__global__
void solver_red_kernel(float *phi, float *epsi,float *epsj, float *epsk, float *repsc, float *rho, float wsor, int xm, int ym, int zm) {

    int xmym = xm * ym;
    int i = 2 * (blockIdx.x * blockDim.x + threadIdx.x) + 1 + xmym;

    if (i < xm * ym * zm + xmym) {
        phi[i] -= wsor * (phi[i] - (epsi[i - 1   ] * phi[i - 1   ] + epsi[i      ] * phi[i + 1   ] +
                                    epsj[i - xm  ] * phi[i - xm  ] + epsj[i      ] * phi[i + xm  ] +
                                    epsk[i - xmym] * phi[i - xmym] + epsk[i      ] * phi[i + xmym] + rho[i - xmym]) * repsc[i - xmym]);
    }

}

__global__
void residue_kernel(float *phi, float *epsi,float *epsj, float *epsk, float *epsc, float *rho, int xm, int ym, int zm, float* res) {

    int xmym = xm * ym;
    int i = (blockIdx.x * blockDim.x + threadIdx.x) + xmym;

    if (i < xm * ym * zm + xmym) {
        res[i - xmym] = epsi[i - 1   ] * phi[i - 1   ] + epsi[i       ] * phi[i + 1   ] +
                        epsj[i - xm  ] * phi[i - xm  ] + epsj[i       ] * phi[i + xm  ] +
                        epsk[i - xmym] * phi[i - xmym] + epsk[i       ] * phi[i + xmym] + rho[i - xmym] - epsc[i - xmym] * phi[i];
    }

}

//*****************************************************
// Recursive V-Cycle
__host__
void VCycle(int level) {

    if (level == MG_NLEVEL - 1) {
        // Solve on coarsest grid
        relax(level, -1);
    } else {
        // Relax & restrict
        if (level < threshold) {
            // On CUDA
            relax_cuda(level, ncyc_before);
            restrict_cuda(level);

        } else {
            // On CPU
            relax(level, ncyc_before);

            restrict_v(16.0, l_rv + mg_index[level], mg_size[level][0], mg_size[level][1], mg_size[level][2], l_bv + mg_index[level + 1], mg_size[level + 1][0], mg_size[level + 1][1], mg_size[level + 1][2]); //bv

        }

        // Reinitialize l_xv on level+1 to zero
        int vnx = mg_size[level + 1][0];
        int vny = mg_size[level + 1][1];
        int vnz = mg_size[level + 1][2];
        int vnxny = vnx * vny;
        int vnxnynz = vnxny * vnz;
        int pa = mg_x_idx[level + 1] + vnxny;
        if (level < threshold - 1) {
            int blocksize = devThreadsPerBlock * devThreadsPerBlock * devThreadsPerBlock;
            int nblocks = (vnxnynz - 1) / blocksize + 1;
            init_vector_kernel<<<nblocks, blocksize>>>(l_xv + pa, vnxnynz);
            cudaLaunchErrorCheck();
            hipDeviceSynchronize();
        } else {
            int pb = pa + vnxnynz;
            for (int i = pa; i < pb; i++) {
                l_xv[i] = 0.0;
            }
        }

        // Recursive call
        VCycle(level + 1);

        // Interpolate & relax
        if (level < threshold) {
            // On CUDA
            interpolate_cuda(level);
            relax_cuda(level, ncyc_after);

            /*@@ Debug of omega
            if (level == 0) {
                relax_cuda(level, 0); // GS
            } else { // level 1, 2
                relax_cuda(level, 2); // SOR, upgoing
            }
            */
        } else {
            // On CPU
            interpolate(level);
            relax(level, ncyc_after);
        }
    }
}

// PB/MG CUDA driver
extern "C"
void pb_mg_cuda_(float *phi_f, float *xs_f) {
    l_itn = 0;

    // Initial norm
    l_inorm = 0.0;
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    cublasErrorCheck(status);
    status = hipblasSasum(handle, l_xmymzm, l_bv, 1, &l_inorm);
    cublasErrorCheck(status);

    bool mgconverged = false;
    while (!mgconverged) {
        for (int i = 0; i < MG_NLEVEL; i++) {
            mg_onorm[i] = 9.9E99;
        }

        l_itn++;

        VCycle(0);

        // Norm
        l_norm = 0.0;
        status = hipblasSasum(handle, l_xmymzm, l_rv, 1, &l_norm);
        cublasErrorCheck(status);

        if (l_itn >= l_maxitn || l_norm <= l_inorm * l_accept ) {
            mgconverged = true;

            if(l_itn >= l_maxitn) {
                printf("PB_MG WARNING: maxitn exceeded (pb_mg_cuda)!\n");
            }
        }
    } // while

    // Destroy context
    hipblasDestroy(handle);

    hipMemcpy(xs_f, l_xv + l_xmym, l_xmymzm * sizeof(float), hipMemcpyDeviceToHost);

    if (l_bcopt != 10 || l_pbkappa != 0) memcpy(phi_f, xs_f, l_xmymzm * sizeof(float));
}

// PB/SOR CUDA driver
extern "C"
void pb_sor_cuda_(float *phi_f, float *xs_f) {
    int threadsPerBlock = 512; // This shouldn't be final, RL

    int nx = l_xm; //mg_size[0][0];
    int ny = l_ym; //mg_size[0][1];
    int nz = l_zm; //mg_size[0][2];
    int nxny = nx * ny;
    int nxnynz = nxny * nz;

    int itn_checknorm = 10;
    float wsor = 1.9;//, wsor1;

    // inverse AD for fast processing later
    int blocks = (nxnynz - 1) / threadsPerBlock + 1;
    inv_vector_kernel<<<blocks, threadsPerBlock>>>(l_ad, l_zv, nxnynz);
    cudaLaunchErrorCheck();

    // initial norm
    float linorm = 0.0;
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    cublasErrorCheck(status);
    status = hipblasSasum(handle, nxnynz, l_bv, 1, &linorm);
    cublasErrorCheck(status);

    bool converged = false;
    int litn = 0;
    float lnorm = 0.0;
    int sblocks = ((int)(nxnynz / 2) + (nxnynz & 1) - 1) / threadsPerBlock + 1;
    while (!converged) {
        solver_red_kernel<<<sblocks, threadsPerBlock>>>(l_xv, l_am1, l_am2, l_am3, l_zv, l_bv, wsor, nx, ny, nz);
        cudaLaunchErrorCheck();
        solver_black_kernel<<<sblocks, threadsPerBlock>>>(l_xv, l_am1, l_am2, l_am3, l_zv, l_bv, wsor, nx, ny, nz);
        cudaLaunchErrorCheck();
        //hipDeviceSynchronize(); // Warning of bug: CUDA_EXCEPTION_15

        litn++;

        // Check convergence
        if (litn % itn_checknorm == 0) {
            // residue
            residue_kernel<<<blocks, threadsPerBlock>>>(l_xv, l_am1, l_am2, l_am3, l_ad, l_bv, nx, ny, nz, l_rv);
            cudaLaunchErrorCheck();

            // norm
            status = hipblasSasum(handle, nxnynz, l_rv, 1, &lnorm);
            cublasErrorCheck(status);

            if (litn >= l_maxitn || lnorm <= l_accept * linorm) {
                converged = true;
                if (litn >= l_maxitn) printf("PB_SOR WARNING: maxitn exceeded (kernel)!\n");
            } // if
        }
    } // while

    // Destroy context
    hipblasDestroy(handle);

    l_itn = litn;
    l_inorm = linorm;
    l_norm = lnorm;

    hipMemcpy(xs_f, l_xv + l_xmym, l_xmymzm * sizeof(float), hipMemcpyDeviceToHost);
    if (l_bcopt != 10 || l_pbkappa != 0) memcpy(phi_f, xs_f, l_xmymzm * sizeof(float));
}

// Return values
extern "C"
int get_itn_() {
    return l_itn;
}

extern "C"
float get_inorm_() {
    return l_inorm;
}

extern "C"
float get_norm_() {
    return l_norm;
}

